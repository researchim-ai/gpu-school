#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>

constexpr int THREADS = 256;

__global__ void vecAdd(const float* a, const float* b, float* c, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) c[idx] = a[idx] + b[idx];
}

void checkCuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char** argv) {
    int n = 1 << 20; // 1M elements
    if (argc == 2) n = std::atoi(argv[1]);

    size_t bytes = n * sizeof(float);

    std::vector<float> h_a(n, 1.0f), h_b(n, 2.0f), h_c(n);

    float *d_a, *d_b, *d_c;
    checkCuda(hipMalloc(&d_a, bytes), "malloc a");
    checkCuda(hipMalloc(&d_b, bytes), "malloc b");
    checkCuda(hipMalloc(&d_c, bytes), "malloc c");

    checkCuda(hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice), "copy a");
    checkCuda(hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice), "copy b");

    int blocks = (n + THREADS - 1) / THREADS;
    vecAdd<<<blocks, THREADS>>>(d_a, d_b, d_c, n);
    checkCuda(hipGetLastError(), "kernel");

    checkCuda(hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost), "copy c back");

    // простая проверка
    bool ok = true;
    for (int i = 0; i < n; ++i) {
        if (fabs(h_c[i] - 3.0f) > 1e-5f) { ok = false; break; }
    }
    printf("VectorAdd %s for %d elements\n", ok ? "PASSED" : "FAILED", n);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return ok ? 0 : 1;
} 