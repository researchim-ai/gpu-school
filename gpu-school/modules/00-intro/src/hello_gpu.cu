#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void hello_kernel() {
    printf("Hello from GPU thread %d\n", threadIdx.x + blockIdx.x * blockDim.x);
}

int main() {
    // Запускаем 1 блок из 32 потоков
    hello_kernel<<<1, 32>>>();

    // Ожидаем завершения GPU
    hipDeviceSynchronize();

    // Проверяем, не возникло ли ошибок при запуске ядра
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        return 1;
    }

    return 0;
} 